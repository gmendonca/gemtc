
#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ void VecSqrt ( void* param1)
{
   // the warp size is hardcoded at 32
   int warp_size = 32;

   // unbox the function parameter
   float* mem = (float*)param1;
   int size = (int)mem[0];
   int As   = (int)mem[1];
   float *A = mem+2;

   // calculate where to store the solution
   float* C = A + As*size;

   //C[tid] = A1[tid] + A2[tid] + A3[tid] + ...;

   // loop over and calculate solution
   int i;
   for(i=0; i<As; i++){
     float * cur = A + i*size;
     int tid = threadIdx.x%warp_size;
     while(tid<size){
       C[tid] += cur[tid];
       tid += warp_size;
     }
   }

   /*   while (tid < size)
   {
     int i, temp;
     temp=0;
     for(i=0; i<As; i++) temp += [tid]);
     C[tid]=temp;
     tid = tid + warp_size;
     }*/
}
